
#include <string.h>
#include <hip/hip_runtime.h>

#include "q6_gpu.h"

int main(int argc, char *argv[]){ 
    srand(SEED);

	// storing times
	class calc_time gpu_t;

	// get from DATABASE stored
	data* tups = retrieve();
    data* device_tups = NULL;

#ifdef DEBUG
	print_data(tups);
#endif


	uint64_t tpch_q6_sum = 0;
    uint64_t *device_q6_sum = NULL;


	uint32_t tests = (argc == 1) ? 1 : atoi(argv[1]);

    for (uint32_t i = 0; i < tests; i++) {
        hipMemcpy((void *) device_q6_sum, &tpch_q6_sum, sizeof(uint64_t*), hipMemcpyHostToDevice);
   
        hipMalloc((void *) device_tups, NUM_TUPLES * sizeof(data));
        // assuming all tuples fit on GPU all at once
        hipMemcpy(device_tups, tups, NUM_TUPLES * sizeof(data), hipMemcpyHostToDevice);

		gpu_t.start();
		q6_gpu(device_tups, device_q6_sum);
		gpu_t.stop();

        hipDeviceSynchronize();
        hipMemcpy(&tpch_q6_sum, device_q6_sum, sizeof(uint64_t*), hipMemcpyDeviceToHost);
	}

	gpu_t.print("CUDA timings", tpch_q6_sum);

	
	free(tups);
	return 0;


}