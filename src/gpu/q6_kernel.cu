#include "hip/hip_runtime.h"
#ifndef _Q6_KERNEL_H_
#define _Q6_KERNEL_H_

#include <stdio.h>
#include "q6_gpu.h"

////////////////////////////////////////////////////////////////////////////////
//! @param lineitem
////////////////////////////////////////////////////////////////////////////////
__global__ void q6_kernel(data *lineitem, uint64_t *sum)
{
	extern __shared__ uint64_t temp[]; 

	uint64_t temp_sum = 0;

	for (int j=threadIdx.x; j < NUM_TUPLES; j += blockDim.x) {
        printf("%d\n", (lineitem + j)->l_shipdate);
	
        if (   ((lineitem + j)->l_shipdate >= Q6_DATE1)   // l_shipdate >= date
            && ((lineitem + j)->l_shipdate < Q6_DATE2)    // l_shipdate < date  
            && ((lineitem + j)->l_discount >= Q6_DISCOUNT1)
            && ((lineitem + j)->l_discount <= Q6_DISCOUNT2)
            && ((lineitem + j)->l_quantity < Q6_QUANTITY)) {
            temp_sum += (lineitem + j)->l_extendedprice * (lineitem + j)->l_discount;
        }
        
	}
    // store local into shared global
	temp[threadIdx.x] = temp_sum;

    // do a reduction here with thread 0
    if (threadIdx.x == 0) {
        for (int j=0; j < blockDim.x; j++) {
            *sum += temp[j];
        }
    }
    // *sum = 132;
}

void q6_gpu_setup(data *tups, data *device_tups, uint64_t *host_q6_sum, uint64_t *device_q6_sum) {
    size_t tableSize = NUM_TUPLES * sizeof(data);
    hipMemcpy((void *) device_q6_sum, host_q6_sum, sizeof(uint64_t*), hipMemcpyHostToDevice);
    hipMalloc((void **) &device_tups, tableSize);
    // assuming all tuples fit on GPU all at once
    hipMemcpy((void *) device_tups, tups, tableSize, hipMemcpyHostToDevice);
}

void q6_gpu_destruct(data *device_tups, uint64_t *host_q6_sum, uint64_t *device_q6_sum) {
    hipDeviceSynchronize();
    hipMemcpy((void *)host_q6_sum, device_q6_sum, sizeof(uint64_t*), hipMemcpyDeviceToHost);
    hipFree(device_tups);
}

void q6_gpu(data *tups, uint64_t *sum) {
    int threadblock_size = BLOCK_SIZE*BLOCK_SIZE;
    int num_blocks = (NUM_TUPLES + threadblock_size - 1) / threadblock_size;

    size_t shared_mem = threadblock_size * sizeof(uint64_t);
    q6_kernel<<<num_blocks, threadblock_size, shared_mem, nullptr>>>(tups, sum);
    // printf("%d\n", sum);
}

#endif 