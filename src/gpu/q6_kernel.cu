#include "hip/hip_runtime.h"
#ifndef _Q6_KERNEL_H_
#define _Q6_KERNEL_H_

#include <stdio.h>
#include "q6_gpu.h"

////////////////////////////////////////////////////////////////////////////////
//! @param lineitem
////////////////////////////////////////////////////////////////////////////////
__global__ void q6_kernel(data *lineitem, uint64_t *sum)
{
	extern __shared__ uint64_t temp[]; 

	uint64_t temp_sum = 0;

	for (int j=threadIdx.x; j < NUM_TUPLES; j += blockDim.x) {
	
        if (   ((lineitem + j)->l_shipdate >= Q6_DATE1)   // l_shipdate >= date
            && ((lineitem + j)->l_shipdate < Q6_DATE2)    // l_shipdate < date  
            && ((lineitem + j)->l_discount >= Q6_DISCOUNT1)
            && ((lineitem + j)->l_discount <= Q6_DISCOUNT2)
            && ((lineitem + j)->l_quantity < Q6_QUANTITY)) {
            temp_sum += (lineitem + j)->l_extendedprice * (lineitem + j)->l_discount;
        }
        
	}
    // store local into shared global
	temp[threadIdx.x] = temp_sum;

    // do a reduction here with thread 0
    if (threadIdx.x == 0) {
        for (int j=0; j < blockDim.x; j++) {
            *sum += temp[j];
        }

    }
}



uint64_t q6_gpu(data *lineitem, uint64_t *sum) {

    int threadblock_size = 256;
    int num_blocks = (NUM_TUPLES + threadblock_size - 1) / threadblock_size;

    size_t shared_mem = threadblock_size * u_int64_t;
    q6_kernel<<<num_blocks, threadblock_size, shared_mem, nullptr>>>(lineitems, sum);
}

#endif 